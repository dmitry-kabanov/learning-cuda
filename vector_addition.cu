
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(float *a, float *b, float *c, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int N = 1024 * 1024;
    size_t size = N * sizeof(float);

    float *ha = (float *) malloc(size);
    float *hb = (float *) malloc(size);
    float *hc = (float *) malloc(size);
    float *hc_check = (float *) malloc(size);

    for (int i = 0; i < N; i++) {
        ha[i] = i;
        hb[i] = i + 1;
        hc_check[i] = ha[i] + hb[i];
    }

    float *da;
    hipMalloc(&da, size);
    float *db;
    hipMalloc(&db, size);
    float *dc;
    hipMalloc(&dc, size);

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);

    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

    int cmp = memcmp(hc_check, hc, size);

    if (cmp == 0) { 
        printf("Arrays are equal.\n");
    } else {
        printf("Arrays are not equal.\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(ha);
    free(hb);
    free(hc);
    
    return 0;
}
