
#include <hip/hip_runtime.h>
#include <stdio.h>

// Matrix is stored as 1d array in row-major order
typedef struct {
    int width;
    int height;
    float *elements;
} Matrix;

#define BLOCK_SIZE 16

#define A_WIDTH  2048
#define A_HEIGHT 2048
#define B_WIDTH  2048
#define B_HEIGHT 2048
#define C_WIDTH  2048
#define C_HEIGHT 2048

__global__ void matmul(const Matrix A, const Matrix B, const Matrix C)
{
    float CValue = 0;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int rowVal = row * A.width;
    for (int k = 0; k < A.width; k++) {
        CValue += A.elements[rowVal + k] * B.elements[k * B.height + col];
    }
    C.elements[row * C.width + col] = CValue;
}

void matmulDriver(const Matrix A, const Matrix B, const Matrix C)
{
    // Load matrix A into device.
    Matrix dA;
    dA.width = A.width;
    dA.height = A.height;
    size_t sizeOfA = A.width * A.height * sizeof(float);
    hipMalloc(&dA.elements, sizeOfA);
    hipMemcpy(dA.elements, A.elements, sizeOfA, hipMemcpyHostToDevice);

    // Load matrix B into device.
    Matrix dB;
    dB.width = B.width;
    dB.height = B.height;
    size_t sizeOfB = B.width * B.height * sizeof(float);
    hipMalloc(&dB.elements, sizeOfB);
    hipMemcpy(dB.elements, B.elements, sizeOfB, hipMemcpyHostToDevice);

    // Allocate matrix C on device.
    Matrix dC;
    dC.width = C.width;
    dC.height = C.height;
    size_t sizeOfC = C.width * C.height * sizeof(float);
    hipMalloc(&dC.elements, sizeOfC);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    matmul<<<dimGrid, dimBlock>>>(A, B, C);

    hipMemcpy(C.elements, dC.elements, sizeOfC, hipMemcpyDeviceToHost);

    hipFree(dA.elements);
    hipFree(dB.elements);
    hipFree(dC.elements);
}

int main()
{
    Matrix A;
    A.width = A_WIDTH;
    A.height = A_HEIGHT;
    size_t sizeOfA = A.width * A.height * sizeof(float);
    A.elements = (float *) malloc(sizeOfA);

    Matrix B;
    B.width = B_WIDTH;
    B.height = B_HEIGHT;
    size_t sizeOfB = B.width * B.height * sizeof(float);
    B.elements = (float *) malloc(sizeOfB);

    Matrix C;
    C.width = C_WIDTH;
    C.height = C_HEIGHT;
    size_t sizeOfC = C.width * C.height * sizeof(float);
    C.elements = (float *) malloc(sizeOfC);

    Matrix C_check;
    C_check.width = C_WIDTH;
    C_check.height = C_HEIGHT;
    C_check.elements = (float *) malloc(sizeOfC);

    for (int i = 0; i < A.height; i++) {
        for (int j = 0; j < A.width; j++) {
            A.elements[i * A.width + j] = i + j;
        }
    }

    for (int i = 0; i < B.height; i++) {
        for (int j = 0; j < B.width; j++) {
            B.elements[i * B.width + j] = i + j;
        }
    }

    int value;
    for (int i = 0; i < C_check.height; i++) {
        for (int j = 0; j < C_check.width; j++) {
            value = 0.0;
            for (int k = 0; k < A.width; k++) {
                value += A.elements[i * A.width + k] * B.elements[k * B.width + j];
            }
            C_check.elements[i * C_check.width + j] = value;
        }
    }
    
    matmulDriver(A, B, C);

    int cmp = memcmp(C_check.elements, C.elements, sizeOfC);

    if (cmp == 0) {
        printf("Arrays are equal.\n");
    } else {
        printf("Arrays are equal.\n");
    }

    return 0;
}
